
#include <hip/hip_runtime.h>
#define THREADS 256
#define WIDTH 16  // shared memory width
#define HEIGHT 16 // shared memory height

#define ETA 0.3      //eta value
#define MOMENTUM 0.3 //momentum value

///////////////////////////////
///////////////////////////////

extern "C" __global__ void backprop2(float *delta,
                                         int hid,
                                         float *ly,
                                         int in,
                                         float *w,
                                         float *oldw) {

    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int index = (hid + 1) * HEIGHT * by + (hid + 1) * ty + tx + 1 + (hid + 1);
    int index_y = HEIGHT * by + ty + 1;
    int index_x = tx + 1;

    w[index] += ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));
    oldw[index] = ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));

    __syncthreads();

    if (ty == 0 && by == 0) {
        w[index_x] += ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
        oldw[index_x] = ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
    }
}
